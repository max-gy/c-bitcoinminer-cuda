#include "hip/hip_runtime.h"
// cd /home/hork/cuda-workspace/CudaSHA256/Debug/files
// time ~/Dropbox/FIIT/APS/Projekt/CpuSHA256/a.out -f ../file-list
// time ../CudaSHA256 -f ../file-list

#include <iostream>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
//#include <hip/hip_runtime.h>
#include "sha256cuda.h"
#include "util.h"
#include "miner.h"
#include <dirent.h>
#include <ctype.h>
#include <chrono>

#include "/usr/local/cuda-11/targets/aarch64-linux/include/hip/hip_runtime.h"
#include "/usr/local/cuda-11/targets/aarch64-linux/include/"


__global__ void sha256_cuda_try(char* a, char* b) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// perform sha256 calculation here
	b[i] = a[i];

	return;
}


void pre_sha256() {
	// compy symbols
//	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}

__global__ void sha256_cuda_hash(uint32_t* result_H) {

	uint32_t K[64] = {0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
	0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
	0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
	0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
	0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
	0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
	0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
	0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
	0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// perform sha256 calculation here

	
	/*
	result_H[i*8+0] = i*8+0;
	result_H[i*8+1] = i*8+1;
	result_H[i*8+2] = i*8+2;
	result_H[i*8+3] = i*8+3;
	result_H[i*8+4] = i*8+4;
	result_H[i*8+5] = i*8+5;
	result_H[i*8+6] = i*8+6;
	result_H[i*8+7] = i*8+7;
*/

	int i = index+1;
	if (i>0) { 
		
		uint32_t a = dev_H[i-1][0];
		uint32_t b = dev_H[i-1][1];
		uint32_t c = dev_H[i-1][2];
		uint32_t d = dev_H[i-1][3];
		uint32_t e = dev_H[i-1][4];
		uint32_t f = dev_H[i-1][5];
		uint32_t g = dev_H[i-1][6];
		uint32_t h = dev_H[i-1][7];

		
		uint32_t W[64];

		for(int j = 0; j < 64; j++)
		{
			uint32_t ch = Ch(e, f, g);
			uint32_t maj = Maj(a, b, c);
			uint32_t Sig0 = Sig0f(a);
			uint32_t Sig1 = Sig1f(e);

			if(j < 16)
				W[j] = dev_M[i-1][j];
			else
				W[j] = sig1(W[j-2]) + W[j-7] + sig0(W[j-15]) + W[j-16];
			
			uint32_t T1 = h + Sig1 + ch + K[j] + W[j];
			uint32_t T2 = Sig0 + maj;
			h = g;
			g = f;
			f = e;
			e = d + T1;
			d = c;
			c = b;
			b = a;
			a = T1 + T2;
		} 

		result_H[i*8+0] = a + dev_H[i-1][0];
		result_H[i*8+1] = b + dev_H[i-1][1];
		result_H[i*8+2] = c + dev_H[i-1][2];
		result_H[i*8+3] = d + dev_H[i-1][3];
		result_H[i*8+4] = e + dev_H[i-1][4];
		result_H[i*8+5] = f + dev_H[i-1][5];
		result_H[i*8+6] = g + dev_H[i-1][6];
		result_H[i*8+7] = h + dev_H[i-1][7];


	}

	//outputlocation[i] = outputlocation[i]*outputlocation[i];

	return;
}

__global__ void sha256_cuda_hash_full(uint64_t * nonce_found) {

	//int index = blockIdx.x * blockDim.x + threadIdx.x;


    //calculates unique thread ID in the block
    int t= (blockDim.x*blockDim.y)*threadIdx.z
		   +(threadIdx.y*blockDim.x)+(threadIdx.x);
	//calculates unique block ID in the grid
    int b= (gridDim.x*gridDim.y)*blockIdx.z
		   +(blockIdx.y*gridDim.x)+(blockIdx.x);
	//block size (this is redundant though) 
	int T= blockDim.x*blockDim.y*blockDim.z;
	//grid size (this is redundant though)
	int B= gridDim.x*gridDim.y*gridDim.z;
	

	int index= b * T + t;

	uint32_t H_0[8] = { 0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a, 0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19 };

	//result_H[index] = index;


	uint32_t blockheader[20];
	for (int bi=0;bi<19;bi++) blockheader[bi] = dev_blockheader[bi];
	blockheader[19] = dev_nonce[0] + index;
    //*(blockheader + 19) = dev_nonce[0] + index;

    //print_bytes((unsigned char*)blockheader, 80);



    for(int i = 0; i < 20; i++)
        blockheader[i] = __Reverse32(blockheader[i]);

	
    uint32_t hash0[8];

	uint32_t outputlocation[8];
	uint32_t H[32][8];

	for (int hash_round = 1;hash_round<3;hash_round++) 
	{
		
		
		int bitlength = 640;
		uint32_t* input = blockheader;
		if (hash_round == 2) {
			bitlength = 256;
			input = outputlocation;
		}	

		

		int wordlength = bitlength / 32 + 1;
		int k = (512 * 512 - bitlength - 1) % 512;
		uint32_t message[10000] = {};

		for(int i = 0; i < wordlength; i++)
			message[i] = input[i];

		if(bitlength % 32 != 0)
			message[bitlength / 32] = message[bitlength / 32] | (1 << (32 - 1 - bitlength % 32));
		else
			message[bitlength / 32] = 1 << 31;

		uint32_t rounds;

		// Assuming our data isn't bigger than 2^32 bits long... which it won't be for a block hash.
		if(wordlength % 16 == 0 || wordlength % 16 == 15)
		{
			message[wordlength + 15 + 16 - wordlength % 16] = bitlength;
			rounds = wordlength / 16 + 2;
		}
		else
		{
			message[wordlength + 15 - wordlength % 16] = bitlength;
			rounds = wordlength / 16 + 1;
		}

		
			
		uint32_t M[32][16];

		for(int i = 0; i < 16; i++)
			for(int j = 0; j <= rounds; j++)
				M[j][i] = message[i + j * 16];


		for(int i = 0; i < 8; i++)
			H[0][i] = H_0[i];


		
		// Here our hash function rounds actually start.
		for(int i = 1; i <= rounds; i++)
		{

			
			uint32_t a = H[i-1][0];
			uint32_t b = H[i-1][1];
			uint32_t c = H[i-1][2];
			uint32_t d = H[i-1][3];
			uint32_t e = H[i-1][4];
			uint32_t f = H[i-1][5];
			uint32_t g = H[i-1][6];
			uint32_t h = H[i-1][7];

			uint32_t W[64];

			
			for(int j = 0; j < 64; j++)
			{
				uint32_t ch = Ch(e, f, g);
				uint32_t maj = Maj(a, b, c);
				uint32_t Sig0 = Sig0f(a);
				uint32_t Sig1 = Sig1f(e);

				if(j < 16)
					W[j] = M[i-1][j];
				else
					W[j] = sig1(W[j-2]) + W[j-7] + sig0(W[j-15]) + W[j-16];
				
				
				uint32_t T1 = h + Sig1 + ch + dev_K[j] + W[j];
				uint32_t T2 = Sig0 + maj;
				h = g;
				g = f;
				f = e;
				e = d + T1;
				d = c;
				c = b;
				b = a;
				a = T1 + T2;
				
			}

			H[i][0] = a + H[i-1][0];
			H[i][1] = b + H[i-1][1];
			H[i][2] = c + H[i-1][2];
			H[i][3] = d + H[i-1][3];
			H[i][4] = e + H[i-1][4];
			H[i][5] = f + H[i-1][5];
			H[i][6] = g + H[i-1][6];
			H[i][7] = h + H[i-1][7];

		
				
		}

		for(int oi = 0; oi < 8; oi++)
			outputlocation[oi] = H[rounds][oi];

		
	}

	nonce_found[0] = 0;
	bool solved = false;
	for(int i = 0; i < 8; i++)
	{
		if(outputlocation[7-i] < dev_difficulty[i])
		{
			solved = true;
			nonce_found[0] = index;
		}
		else if(outputlocation[7-i] > dev_difficulty[i])
			break;
		// And if they're equal, we keep going!
	}
	nonce_found[0] = index;
//	result_H[index] = solved;

	return;

 
}


void __hashblock(uint32_t _nonce, char* version, char* prevhash, 
	char* merkle_root, char* time, char* nbits)
{


    uint32_t K[64] = {0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
		0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
		0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
		0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
		0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
		0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
		0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
		0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
		0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
		0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
		0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
		0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
		0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
		0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
		0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
		0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};
		
	

	uint32_t nonce[1];
	nonce[0] = _nonce;

	uint32_t difficulty[8];
    uint32_t bits[1];
    hexstr_to_intarray(nbits, bits);
    bits_to_difficulty(*bits, difficulty);

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_K), K, sizeof(K), 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_difficulty), difficulty, sizeof(difficulty), 0, hipMemcpyHostToDevice));

	int solved = 0;
	while (!solved) {

		uint32_t blockheader[20];

		hexstr_to_intarray(version, blockheader);
		hexstr_to_intarray(prevhash, blockheader + 1);
		hexstr_to_intarray(merkle_root, blockheader + 9);
		hexstr_to_intarray(time, blockheader + 17);
		hexstr_to_intarray(nbits, blockheader + 18);
		*(blockheader + 19) = nonce[0];

		const uint64_t SIZE_BUFFER = 1024*1024;
		uint64_t * cuda_result_nonce_1 = 0;
		uint64_t * cuda_result_nonce_2 = 0;

		uint64_t result_nonce[1];


		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_nonce), nonce, sizeof(nonce), 0, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_blockheader), blockheader, sizeof(blockheader), 0, hipMemcpyHostToDevice));

		std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();


		hipMalloc(&cuda_result_nonce_1, sizeof(result_nonce));

		sha256_cuda_hash_full <<< 1024, 1024 >>> (cuda_result_nonce_1);

		//hipMalloc(&cuda_result_nonce_2, sizeof(result_nonce));

		//sha256_cuda_hash_full <<< (64,64,64), 1024 >>> (cuda_result_nonce_2);

		
		hipMemcpy(result_nonce,cuda_result_nonce_1,sizeof(result_nonce),hipMemcpyDeviceToHost); 

		nonce[0]+=result_nonce[0];

		std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
		uint64_t duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
		uint64_t hashrate = result_nonce[0] / ((uint64_t)duration);
		std::cout << "Currently mining at " << hashrate << "000 hashes / second" << std::endl;
			
		
		std::cout << "finished cuda \n";
		std::cout << result_nonce[0] << " ";// << nonce[0] << "\n";
		

	}



}

void run_hash (uint32_t * input, int bitlength, uint32_t * outputlocation) {

	uint32_t H_0[8] = { 0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a, 0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19 };

    int wordlength = bitlength / 32 + 1;
    int k = (512 * 512 - bitlength - 1) % 512;
    uint32_t message[10000] = {};


    for(int i = 0; i < wordlength; i++)
        message[i] = input[i];

    if(bitlength % 32 != 0)
        message[bitlength / 32] = message[bitlength / 32] | (1 << (32 - 1 - bitlength % 32));
    else
        message[bitlength / 32] = 1 << 31;
    
    uint32_t rounds;

    // Assuming our data isn't bigger than 2^32 bits long... which it won't be for a block hash.
    if(wordlength % 16 == 0 || wordlength % 16 == 15)
    {
        message[wordlength + 15 + 16 - wordlength % 16] = bitlength;
        rounds = wordlength / 16 + 2;
    }
    else
    {
        message[wordlength + 15 - wordlength % 16] = bitlength;
        rounds = wordlength / 16 + 1;
	}
	
	uint32_t M[32][16];

    for(int i = 0; i < 16; i++)
        for(int j = 0; j <= rounds; j++)
			M[j][i] = message[i + j * 16];
			

	uint32_t H[32][8];
	
    for(int i = 0; i < 8; i++)
        H[0][i] = H_0[i];
    

	
	//cudahash(H, M, rounds, outputlocation);

	uint32_t* cuda_result = 0;

	uint32_t* cuda_message= 0;

	//checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_message), message, sizeof(message), 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_H), H, sizeof(H), 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_M), M, sizeof(M), 0, hipMemcpyHostToDevice));

	hipMallocManaged(&cuda_result, 32*8*sizeof(uint32_t));

	hipMemcpy(cuda_message,message,sizeof(M),hipMemcpyHostToDevice); 

	sha256_cuda_hash <<< 1, rounds >>> (cuda_result);

	uint32_t __H[32*8];

	hipMemcpy(__H,cuda_result,sizeof(__H),hipMemcpyDeviceToHost); 


	for(int i = 0; i < 8; i++)
		__H[i] = H_0[i];
		

	for(int i = 0; i < 8; i++)
		outputlocation[i] = __H[rounds*8+i];

    //std::cout << "\n Run hash input\n";
	//for (int y=0;y<32*8;y++) 
        	//std::cout << __H[y] << " ";
    
}


void runJobs_workingedition(JOB ** jobs, int n){

	char a[4] = "sav";
	char b[sizeof(a)/sizeof(char)];
	char *ca = 0;
	char *cb = 0;

	

	hipMalloc(&ca, sizeof(a));
	hipMalloc(&cb, sizeof(b));
	hipMemcpy(ca, a, sizeof(a), hipMemcpyHostToDevice);

	
	sha256_cuda_try <<< 1, sizeof(a)/sizeof(char) >>> (ca, cb);

	hipMemcpy(b, cb, sizeof(b), hipMemcpyDeviceToHost);

	std::cout << "hhelo";
	std::cout << b;
	std::cout << 10;
}

