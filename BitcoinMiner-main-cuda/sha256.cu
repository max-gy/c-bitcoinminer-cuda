#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <dirent.h>
#include <ctype.h>


#include "/usr/local/cuda-11.4/targets/aarch64-linux/include/hip/hip_runtime.h"
#include "/usr/local/cuda-11.4/targets/aarch64-linux/include/"



uint32_t rotateInt(uint32_t inputWord, int numberOfBitsToRotate) 
{
    int bitWidth = sizeof(inputWord) * 8;
    // Rotating 32 bits on a 32-bit integer is the same as rotating 0 bits;
    //   33 bits -> 1 bit; etc.
    numberOfBitsToRotate = numberOfBitsToRotate % bitWidth;

    uint32_t tempWord = inputWord;

    // Rotate input to the right
    inputWord = inputWord >> numberOfBitsToRotate;

    // Build mask for carried over bits
    tempWord = tempWord << (bitWidth - numberOfBitsToRotate);

    return inputWord | tempWord;
}

int Ch(int x, int y, int z)
{
    return ((x & y) ^ (~x & z));
}

int Maj(int x, int y, int z)
{
    return ((x & y) ^ (x & z) ^ (y & z));
}

int Sig0f(int x)
{
    return(rotateInt(x, 2) ^ rotateInt(x, 13) ^ rotateInt(x, 22));
}

int Sig1f(int x)
{
    return(rotateInt(x, 6) ^ rotateInt(x, 11) ^ rotateInt(x,25));
}

uint32_t sig0(uint32_t x)
{
    return(rotateInt(x, 7) ^ rotateInt(x, 18) ^ (x >> 3));
}

uint32_t sig1(uint32_t x)
{
    return(rotateInt(x, 17) ^ rotateInt(x, 19) ^ (x >> 10));
}

void hash(uint32_t *input, int bitlength, uint32_t *outputlocation)
{
    uint32_t H_0[8] = { 0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a, 0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19 };

    uint32_t K[64] = {0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

    int wordlength = bitlength / 32 + 1;
    int k = (512 * 512 - bitlength - 1) % 512;
    uint32_t message[10000] = {};

    for(int i = 0; i < wordlength; i++)
        message[i] = input[i];

    if(bitlength % 32 != 0)
        message[bitlength / 32] = message[bitlength / 32] | (1 << (32 - 1 - bitlength % 32));
    else
        message[bitlength / 32] = 1 << 31;
    
    uint32_t rounds;

    // Assuming our data isn't bigger than 2^32 bits long... which it won't be for a block hash.
    if(wordlength % 16 == 0 || wordlength % 16 == 15)
    {
        message[wordlength + 15 + 16 - wordlength % 16] = bitlength;
        rounds = wordlength / 16 + 2;
    }
    else
    {
        message[wordlength + 15 - wordlength % 16] = bitlength;
        rounds = wordlength / 16 + 1;
    }
        
    uint32_t M[32][16];

    for(int i = 0; i < 16; i++)
        for(int j = 0; j <= rounds; j++)
            M[j][i] = message[i + j * 16];
    
    uint32_t H[32][8];

    for(int i = 0; i < 8; i++)
        H[0][i] = H_0[i];

    // Here our hash function rounds actually start.
    for(int i = 1; i <= rounds; i++)
    {
        uint32_t a = H[i-1][0];
        uint32_t b = H[i-1][1];
        uint32_t c = H[i-1][2];
        uint32_t d = H[i-1][3];
        uint32_t e = H[i-1][4];
        uint32_t f = H[i-1][5];
        uint32_t g = H[i-1][6];
        uint32_t h = H[i-1][7];

        uint32_t W[64];

        for(int j = 0; j < 64; j++)
        {
            uint32_t ch = Ch(e, f, g);
            uint32_t maj = Maj(a, b, c);
            uint32_t Sig0 = Sig0f(a);
            uint32_t Sig1 = Sig1f(e);

            if(j < 16)
                W[j] = M[i-1][j];
            else
                W[j] = sig1(W[j-2]) + W[j-7] + sig0(W[j-15]) + W[j-16];
            
            uint32_t T1 = h + Sig1 + ch + K[j] + W[j];
            uint32_t T2 = Sig0 + maj;
            h = g;
            g = f;
            f = e;
            e = d + T1;
            d = c;
            c = b;
            b = a;
            a = T1 + T2;
        }

        H[i][0] = a + H[i-1][0];
        H[i][1] = b + H[i-1][1];
        H[i][2] = c + H[i-1][2];
        H[i][3] = d + H[i-1][3];
        H[i][4] = e + H[i-1][4];
        H[i][5] = f + H[i-1][5];
        H[i][6] = g + H[i-1][6];
        H[i][7] = h + H[i-1][7];
    }

    for(int i = 0; i < 8; i++)
        outputlocation[i] = H[rounds][i];
}

// CUDA PART

#define checkCudaErrors(x) \
{ \
    hipGetLastError(); \
    x; \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) \
        printf("GPU: hipError_t %d (%s)\n", err, hipGetErrorString(err)); \
}

#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#define ROTRIGHT(a,b) (((a) >> (b)) | ((a) << (32-(b))))

#define CH(x,y,z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x,y,z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x,2) ^ ROTRIGHT(x,13) ^ ROTRIGHT(x,22))
#define EP1(x) (ROTRIGHT(x,6) ^ ROTRIGHT(x,11) ^ ROTRIGHT(x,25))
#define SIG0(x) (ROTRIGHT(x,7) ^ ROTRIGHT(x,18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x,17) ^ ROTRIGHT(x,19) ^ ((x) >> 10))


typedef unsigned char BYTE;             // 8-bit byte
typedef uint32_t  WORD;             // 32-bit word, change to "long" for 16-bit machines

typedef struct JOB {
	BYTE * data;
	unsigned long long size;
	BYTE digest[64];
	char fname[128];
}JOB;


typedef struct {
	BYTE data[64];
	WORD datalen;
	unsigned long long bitlen;
	WORD state[8];
} SHA256_CTX;

__constant__ WORD dev_k[64];

static const WORD host_k[64] = {
	0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
	0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
	0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
	0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
	0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
	0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
	0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
	0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

/*********************** FUNCTION DECLARATIONS **********************/
char * print_sha(BYTE * buff);
__device__ void sha256_init(SHA256_CTX *ctx);
__device__ void sha256_update(SHA256_CTX *ctx, const BYTE data[], size_t len);
__device__ void sha256_final(SHA256_CTX *ctx, BYTE hash[]);


char * hash_to_string(BYTE * buff) {
	char * string = (char *)malloc(70);
	int k, i;
	for (i = 0, k = 0; i < 32; i++, k+= 2)
	{
		sprintf(string + k, "%.2x", buff[i]);
		//printf("%02x", buff[i]);
	}
	string[64] = 0;
	return string;
}

void print_job(JOB * j){
	printf("%s  %s\n", hash_to_string(j->digest), j->fname);
}

void print_jobs(JOB ** jobs, int n) {
	for (int i = 0; i < n; i++)
	{
        print_job(jobs[i]);
		// printf("@ %p JOB[%i] \n", jobs[i], i);
		// printf("\t @ 0x%p data = %x \n", jobs[i]->data, (jobs[i]->data == 0)? 0 : jobs[i]->data[0]);
		// printf("\t @ 0x%p size = %llu \n", &(jobs[i]->size), jobs[i]->size);
		// printf("\t @ 0x%p fname = %s \n", &(jobs[i]->fname), jobs[i]->fname);
		// printf("\t @ 0x%p digest = %s \n------\n", jobs[i]->digest, hash_to_string(jobs[i]->digest));
	}
}

__device__ void mycpy12(uint32_t *d, const uint32_t *s) {
#pragma unroll 3
    for (int k=0; k < 3; k++) d[k] = s[k];
}

__device__ void mycpy16(uint32_t *d, const uint32_t *s) {
#pragma unroll 4
    for (int k=0; k < 4; k++) d[k] = s[k];
}

__device__ void mycpy32(uint32_t *d, const uint32_t *s) {
#pragma unroll 8
    for (int k=0; k < 8; k++) d[k] = s[k];
}

__device__ void mycpy44(uint32_t *d, const uint32_t *s) {
#pragma unroll 11
    for (int k=0; k < 11; k++) d[k] = s[k];
}

__device__ void mycpy48(uint32_t *d, const uint32_t *s) {
#pragma unroll 12
    for (int k=0; k < 12; k++) d[k] = s[k];
}

__device__ void mycpy64(uint32_t *d, const uint32_t *s) {
#pragma unroll 16
    for (int k=0; k < 16; k++) d[k] = s[k];
}

__device__ void sha256_transform(SHA256_CTX *ctx, const BYTE data[])
{
	WORD a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];
    WORD S[8];

    //mycpy32(S, ctx->state);

    #pragma unroll 16
	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);

    #pragma unroll 64
	for (; i < 64; ++i)
		m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];
	f = ctx->state[5];
	g = ctx->state[6];
	h = ctx->state[7];

    #pragma unroll 64
	for (i = 0; i < 64; ++i) {
		t1 = h + EP1(e) + CH(e, f, g) + dev_k[i] + m[i];
		t2 = EP0(a) + MAJ(a, b, c);
		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}

	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
	ctx->state[5] += f;
	ctx->state[6] += g;
	ctx->state[7] += h;
}

__device__ void sha256_init(SHA256_CTX *ctx)
{
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x6a09e667;
	ctx->state[1] = 0xbb67ae85;
	ctx->state[2] = 0x3c6ef372;
	ctx->state[3] = 0xa54ff53a;
	ctx->state[4] = 0x510e527f;
	ctx->state[5] = 0x9b05688c;
	ctx->state[6] = 0x1f83d9ab;
	ctx->state[7] = 0x5be0cd19;
}

__device__ void sha256_update(SHA256_CTX *ctx, const BYTE data[], size_t len)
{
	WORD i;

	// for each byte in message
	for (i = 0; i < len; ++i) {
		// ctx->data == message 512 bit chunk
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;
		if (ctx->datalen == 64) {
			sha256_transform(ctx, ctx->data);
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

__device__ void sha256_final(SHA256_CTX *ctx, BYTE hash[])
{
	WORD i;

	i = ctx->datalen;

	// Pad whatever data is left in the buffer.
	if (ctx->datalen < 56) {
		ctx->data[i++] = 0x80;
		while (i < 56)
			ctx->data[i++] = 0x00;
	}
	else {
		ctx->data[i++] = 0x80;
		while (i < 64)
			ctx->data[i++] = 0x00;
		sha256_transform(ctx, ctx->data);
		memset(ctx->data, 0, 56);
	}

	// Append to the padding the total message's length in bits and transform.
	ctx->bitlen += ctx->datalen * 8;
	ctx->data[63] = ctx->bitlen;
	ctx->data[62] = ctx->bitlen >> 8;
	ctx->data[61] = ctx->bitlen >> 16;
	ctx->data[60] = ctx->bitlen >> 24;
	ctx->data[59] = ctx->bitlen >> 32;
	ctx->data[58] = ctx->bitlen >> 40;
	ctx->data[57] = ctx->bitlen >> 48;
	ctx->data[56] = ctx->bitlen >> 56;
	sha256_transform(ctx, ctx->data);

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash.
	for (i = 0; i < 4; ++i) {
		hash[i] = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4] = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8] = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 20] = (ctx->state[5] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 24] = (ctx->state[6] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 28] = (ctx->state[7] >> (24 - i * 8)) & 0x000000ff;
	}

}

__global__ void sha256_cuda(JOB ** jobs, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// perform sha256 calculation here
	if (i < n){
		SHA256_CTX ctx;
		sha256_init(&ctx);
		sha256_update(&ctx, jobs[i]->data, jobs[i]->size);
		sha256_final(&ctx, jobs[i]->digest);
	}
}



void runJobs(JOB ** jobs, int n){
	int blockSize = 4;
	int numBlocks = (n + blockSize - 1) / blockSize;
	sha256_cuda <<< numBlocks, blockSize >>> (jobs, n);
}


JOB * JOB_init(BYTE * data, long size, char * fname) {
	JOB * j;
	checkCudaErrors(hipMallocManaged(&j, sizeof(JOB)));	//j = (JOB *)malloc(sizeof(JOB));
	checkCudaErrors(hipMallocManaged(&(j->data), size));
	j->data = data;
	j->size = size;
	for (int i = 0; i < 64; i++)
	{
		j->digest[i] = 0xff;
	}
	std::cout << "JOB_init inside" << std::endl;
	strcpy(j->fname, fname);
	return j;
}

unsigned char *uint32_to_char_array(uint32_t n)
{
    unsigned char *a;

   // a = wrap_calloc(4, sizeof(unsigned char));

    a[0] = (n >> 24) & 0xff;  /* high-order (leftmost) byte: bits 24-31 */
    a[1] = (n >> 16) & 0xff;  /* next byte, counting from left: bits 16-23 */
    a[2] = (n >>  8) & 0xff;  /* next byte, bits 8-15 */
    a[3] = n         & 0xff;  /* low-order byte: bits 0-7 */

    return a;
}

void hash_CUDA (uint32_t input, unsigned long bitlength, uint32_t * outputlocation) {
    //    checkCudaErrors(hipMallocManaged(&jobs, n * sizeof(JOB *)));

		// iterate over file list - non optional arguments
			BYTE * inputBytes[4];
			char * outputlocationBytes[4];

			//uint32_to_char_array(input);
			uint8_t byteval[4];
			
			for(int i = 0; i < 4; i++) byteval[i] = input >> (i*8);
			for(int i = 0; i < 4; i++) byteval[i] = input >> (i*8);


			std::cout << "JOB_init 0" << input << std::endl;

			//for (int i=0; i<4 ;++i)
			//	inputBytes[i] = &((BYTE*)*input)[3-i];

			JOB * jobs[4];

			std::cout << "JOB_init 1" << byteval[0] << std::endl;

			jobs[0] = JOB_init(&byteval[0], bitlength, outputlocationBytes[0]);

			std::cout << "JOB_init 2" << std::endl;

			jobs[1] = JOB_init(&byteval[1], bitlength, outputlocationBytes[1]);
			std::cout << "JOB_init 3" << std::endl;

			jobs[2] = JOB_init(&byteval[2], bitlength, outputlocationBytes[2]);
			std::cout << "JOB_init 4" << std::endl;

			jobs[3] = JOB_init(&byteval[3], bitlength, outputlocationBytes[3]);

			std::cout << "JOB_init after" << std::endl;

			
			outputlocation = ((uint32_t*) &outputlocationBytes);

			std::cout << "Before runJobs " << std::endl;

//        }

       // pre_sha256();
        runJobs(jobs, 1);
  //  }
}
   	

