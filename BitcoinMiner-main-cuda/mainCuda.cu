#include "hip/hip_runtime.h"
// cd /home/hork/cuda-workspace/CudaSHA256/Debug/files
// time ~/Dropbox/FIIT/APS/Projekt/CpuSHA256/a.out -f ../file-list
// time ../CudaSHA256 -f ../file-list

#include <iostream>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
//#include <hip/hip_runtime.h>
#include "sha256cuda.h"

#include <dirent.h>
#include <ctype.h>

#include "/usr/local/cuda-11.4/targets/aarch64-linux/include/hip/hip_runtime.h"
#include "/usr/local/cuda-11.4/targets/aarch64-linux/include/"

char * trim(char *str){
    size_t len = 0;
    char *frontp = str;
    char *endp = NULL;

    if( str == NULL ) { return NULL; }
    if( str[0] == '\0' ) { return str; }

    len = strlen(str);
    endp = str + len;

    /* Move the front and back pointers to address the first non-whitespace
     * characters from each end.
     */
    while( isspace((unsigned char) *frontp) ) { ++frontp; }
    if( endp != frontp )
    {
        while( isspace((unsigned char) *(--endp)) && endp != frontp ) {}
    }

    if( str + len - 1 != endp )
            *(endp + 1) = '\0';
    else if( frontp != str &&  endp == frontp )
            *str = '\0';

    /* Shift the string so that it starts at str so that if it's dynamically
     * allocated, we can still free it on the returned pointer.  Note the reuse
     * of endp to mean the front of the string buffer now.
     */
    endp = str;
    if( frontp != str )
    {
            while( *frontp ) { *endp++ = *frontp++; }
            *endp = '\0';
    }


    return str;
}

__global__ void sha256_cuda(JOB * jobs, JOB * results) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// perform sha256 calculation here

	//results[i].data = jobs[i].data; 
	if (0 < 1){
		SHA256_CTX ctx;
	}

	 //reinterpret_cast<unsigned char*>('2');
	return;
}

__global__ void sha256_cuda_try(char* a, char* b) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// perform sha256 calculation here
	b[i] = a[i];

	return;
}

__global__ void sha256_cuda_hash_640(uint32_t *input, uint32_t *outputlocation) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// perform sha256 calculation here
	cudahash(&input[i], 640, &outputlocation[i]);

	//outputlocation[i] = outputlocation[i]*outputlocation[i];

	return;
}

__global__ void sha256_cuda_hash_256(uint32_t *input, uint32_t *outputlocation) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// perform sha256 calculation here
	cudahash(&input[i], 256, &outputlocation[i]);

	//outputlocation[i] = outputlocation[i]*outputlocation[i];

	return;
}



void pre_sha256() {
	// compy symbols
//	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}


void runJobs(JOB ** jobs, int n){
	int blockSize = 4;
	int numBlocks = (n + blockSize - 1) / blockSize;

	JOB* cuda_jobs = 0;
	JOB* cuda_jobs_r = 0;

	BYTE * buff;

	
	JOB js[1];
	char a[9] = "hellohey";
	unsigned char* d = reinterpret_cast<unsigned char*>(a);
	buff = reinterpret_cast<unsigned char*>(a);
	
	std::cout << "Init Job";
	//checkCudaErrors(hipMallocManaged(&js, sizeof(JOB)));	//j = (JOB *)malloc(sizeof(JOB));
	//checkCudaErrors(hipMallocManaged(&(js.data), sizeof(a)));
	js[0].data = buff;
	js[0].size = sizeof(a);
	for (int i = 0; i < 64; i++)
	{
		js[0].digest[i] = 0xff;
	}
	strcpy(js[0].fname, "1");

	JOB jsr[sizeof(js)/sizeof(JOB)];

	checkCudaErrors(hipMalloc(&cuda_jobs, sizeof(js)));
	checkCudaErrors(hipMalloc(&cuda_jobs_r, sizeof(jsr)));

	checkCudaErrors(hipMemcpy(cuda_jobs, js, sizeof(js), hipMemcpyHostToDevice));
	
	sha256_cuda <<< 1, sizeof(js)/sizeof(JOB) >>> (cuda_jobs, cuda_jobs_r);

	checkCudaErrors(hipMemcpy(jsr, cuda_jobs_r, sizeof(jsr), hipMemcpyDeviceToHost));
	
	std::cout << jsr[0].digest;
	//print_jobs(js, 1);

}

void run_hash (uint32_t * input, int bitlength, uint32_t * outputlocation) {


	uint32_t *inputCuda = 0;
	uint32_t *outputlocationCuda = 0;

	hipMalloc(&inputCuda, sizeof(uint32_t)*20);
	hipMalloc(&outputlocationCuda, sizeof(uint32_t)*8);

	hipMemcpy(inputCuda, input, sizeof(uint32_t)*20, hipMemcpyHostToDevice);

	if (bitlength == 640) sha256_cuda_hash_640 <<< 1, sizeof(uint32_t)*8 >>> (inputCuda, outputlocationCuda);

	if (bitlength == 256) sha256_cuda_hash_256 <<< 1, sizeof(outputlocation) >>> (inputCuda, outputlocationCuda);

	hipMemcpy(outputlocation, outputlocationCuda, sizeof(outputlocation), hipMemcpyDeviceToHost);


	//std::cout << &input;
	//std::cout << &outputlocation;
}


void runJobs_workingedition(JOB ** jobs, int n){

	char a[4] = "sav";
	char b[sizeof(a)/sizeof(char)];
	char *ca = 0;
	char *cb = 0;

	hipMalloc(&ca, sizeof(a));
	hipMalloc(&cb, sizeof(b));
	hipMemcpy(ca, a, sizeof(a), hipMemcpyHostToDevice);

	sha256_cuda_try <<< 1, sizeof(a)/sizeof(char) >>> (ca, cb);

	hipMemcpy(b, cb, sizeof(b), hipMemcpyDeviceToHost);

	std::cout << "hhelo";
	std::cout << b;
	std::cout << 10;
}



JOB * JOB_init(BYTE * data, long size, char * fname) {
	JOB * j;
	std::cout << "Init Job";
	checkCudaErrors(hipMallocManaged(&j, sizeof(JOB)));	//j = (JOB *)malloc(sizeof(JOB));
	checkCudaErrors(hipMallocManaged(&(j->data), size));
	j->data = data;
	j->size = size;
	for (int i = 0; i < 64; i++)
	{
		j->digest[i] = 0xff;
	}
	strcpy(j->fname, fname);
	return j;
}

int buildCUDAJobs(ssize_t hash) {

	std::cout << "JOB???";

	BYTE * buff;
	JOB ** jobs;

	//unsigned char myString [] = "This is my string";
	//BYTE * byte = &myString[0];
	checkCudaErrors(hipMallocManaged(&jobs, 1 * sizeof(JOB *)));

	char a[9] = "hellohey";
	unsigned char* d = reinterpret_cast<unsigned char*>(a);
	buff = reinterpret_cast<unsigned char*>(a);
	
	jobs[0] = JOB_init(buff, sizeof(a), "1");
	
	//print_jobs(jobs, 1);


	pre_sha256();
	
	std::cout << "run JOB???";

	runJobs(jobs, 1);

	std::cout << "after run JOB???";


	hipDeviceSynchronize();
	hipDeviceReset();
	return 10;
}