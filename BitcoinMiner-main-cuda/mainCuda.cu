#include "hip/hip_runtime.h"
// cd /home/hork/cuda-workspace/CudaSHA256/Debug/files
// time ~/Dropbox/FIIT/APS/Projekt/CpuSHA256/a.out -f ../file-list
// time ../CudaSHA256 -f ../file-list

#include <iostream>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
//#include <hip/hip_runtime.h>
#include "sha256cuda.h"

#include <dirent.h>
#include <ctype.h>

#include "/usr/local/cuda-11.4/targets/aarch64-linux/include/hip/hip_runtime.h"
#include "/usr/local/cuda-11.4/targets/aarch64-linux/include/"


__global__ void sha256_cuda_try(char* a, char* b) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// perform sha256 calculation here
	b[i] = a[i];

	return;
}


void pre_sha256() {
	// compy symbols
//	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}

__global__ void sha256_cuda_hash(uint32_t* result_H) {

	uint32_t K[64] = {0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
	0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
	0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
	0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
	0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
	0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
	0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
	0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
	0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// perform sha256 calculation here

	
	/*
	result_H[i*8+0] = i*8+0;
	result_H[i*8+1] = i*8+1;
	result_H[i*8+2] = i*8+2;
	result_H[i*8+3] = i*8+3;
	result_H[i*8+4] = i*8+4;
	result_H[i*8+5] = i*8+5;
	result_H[i*8+6] = i*8+6;
	result_H[i*8+7] = i*8+7;
*/

	int i = index+1;
	if (i>0) { 
		
		uint32_t a = dev_H[i-1][0];
		uint32_t b = dev_H[i-1][1];
		uint32_t c = dev_H[i-1][2];
		uint32_t d = dev_H[i-1][3];
		uint32_t e = dev_H[i-1][4];
		uint32_t f = dev_H[i-1][5];
		uint32_t g = dev_H[i-1][6];
		uint32_t h = dev_H[i-1][7];

		
		uint32_t W[64];

		for(int j = 0; j < 64; j++)
		{
			uint32_t ch = Ch(e, f, g);
			uint32_t maj = Maj(a, b, c);
			uint32_t Sig0 = Sig0f(a);
			uint32_t Sig1 = Sig1f(e);

			if(j < 16)
				W[j] = dev_M[i-1][j];
			else
				W[j] = sig1(W[j-2]) + W[j-7] + sig0(W[j-15]) + W[j-16];
			
			uint32_t T1 = h + Sig1 + ch + K[j] + W[j];
			uint32_t T2 = Sig0 + maj;
			h = g;
			g = f;
			f = e;
			e = d + T1;
			d = c;
			c = b;
			b = a;
			a = T1 + T2;
		} 

		result_H[i*8+0] = a + dev_H[i-1][0];
		result_H[i*8+1] = b + dev_H[i-1][1];
		result_H[i*8+2] = c + dev_H[i-1][2];
		result_H[i*8+3] = d + dev_H[i-1][3];
		result_H[i*8+4] = e + dev_H[i-1][4];
		result_H[i*8+5] = f + dev_H[i-1][5];
		result_H[i*8+6] = g + dev_H[i-1][6];
		result_H[i*8+7] = h + dev_H[i-1][7];


	}

	//outputlocation[i] = outputlocation[i]*outputlocation[i];

	return;
}

void hashblock(uint32_t nonce, char* version, char* prevhash, 
	char* merkle_root, char* time, char* nbits, uint32_t* result)
{


}

void run_hash (uint32_t * input, int bitlength, uint32_t * outputlocation) {

	uint32_t H_0[8] = { 0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a, 0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19 };

    int wordlength = bitlength / 32 + 1;
    int k = (512 * 512 - bitlength - 1) % 512;
    uint32_t message[10000] = {};


    for(int i = 0; i < wordlength; i++)
        message[i] = input[i];

    if(bitlength % 32 != 0)
        message[bitlength / 32] = message[bitlength / 32] | (1 << (32 - 1 - bitlength % 32));
    else
        message[bitlength / 32] = 1 << 31;
    
    uint32_t rounds;

    // Assuming our data isn't bigger than 2^32 bits long... which it won't be for a block hash.
    if(wordlength % 16 == 0 || wordlength % 16 == 15)
    {
        message[wordlength + 15 + 16 - wordlength % 16] = bitlength;
        rounds = wordlength / 16 + 2;
    }
    else
    {
        message[wordlength + 15 - wordlength % 16] = bitlength;
        rounds = wordlength / 16 + 1;
	}
	
	uint32_t M[32][16];

    for(int i = 0; i < 16; i++)
        for(int j = 0; j <= rounds; j++)
			M[j][i] = message[i + j * 16];
			

	uint32_t H[32][8];
	
    for(int i = 0; i < 8; i++)
        H[0][i] = H_0[i];
    

	
	//cudahash(H, M, rounds, outputlocation);

	uint32_t* cuda_result = 0;

	uint32_t* cuda_message= 0;

	//checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_message), message, sizeof(message), 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_H), H, sizeof(H), 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_M), M, sizeof(M), 0, hipMemcpyHostToDevice));

	hipMallocManaged(&cuda_result, 32*8*sizeof(uint32_t));

	hipMemcpy(cuda_message,message,sizeof(M),hipMemcpyHostToDevice); 

	sha256_cuda_hash <<< 1, rounds >>> (cuda_result);

	uint32_t __H[32*8];

	hipMemcpy(__H,cuda_result,sizeof(__H),hipMemcpyDeviceToHost); 


	for(int i = 0; i < 8; i++)
		__H[i] = H_0[i];
		

	for(int i = 0; i < 8; i++)
		outputlocation[i] = __H[rounds*8+i];

    //std::cout << "\n Run hash input\n";
	//for (int y=0;y<32*8;y++) 
        	//std::cout << __H[y] << " ";
    
}


void runJobs_workingedition(JOB ** jobs, int n){

	char a[4] = "sav";
	char b[sizeof(a)/sizeof(char)];
	char *ca = 0;
	char *cb = 0;

	

	hipMalloc(&ca, sizeof(a));
	hipMalloc(&cb, sizeof(b));
	hipMemcpy(ca, a, sizeof(a), hipMemcpyHostToDevice);

	
	sha256_cuda_try <<< 1, sizeof(a)/sizeof(char) >>> (ca, cb);

	hipMemcpy(b, cb, sizeof(b), hipMemcpyDeviceToHost);

	std::cout << "hhelo";
	std::cout << b;
	std::cout << 10;
}

